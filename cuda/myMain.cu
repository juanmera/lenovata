#include <stdio.h>
#include <stdlib.h>
#include "mySha256.cuh"

int main(void) {
    uint HostPasswordSize = sizeof(Password);
    uint HostHashSize = sizeof(Hash);

    Password *pwd, *cudaPwd;
    Hash *h, *cudaH;

    hipHostAlloc((void **) &pwd, HostPasswordSize, hipHostMallocDefault);
    hipHostAlloc((void **) &h, HostHashSize, hipHostMallocDefault);
    // malloc(&h, HostHashSize);
    hipMalloc(&cudaPwd, HostPasswordSize);
    hipMalloc(&cudaH, HostHashSize);
    pwd->p[0] = 'a';
    pwd->l = 1;

    hipMemcpy(cudaPwd, pwd, HostPasswordSize, hipMemcpyHostToDevice);
    sha256Digest(cudaPwd, cudaH);
    hipMemcpy(h, cudaH, HostHashSize, hipMemcpyDeviceToHost);
    for (int i=0; i<32; ++i) {
        printf("%02x ", h->hb[i]);
    }
    printf("\n");
    for (int i=0; i<8; ++i) {
        printf("%08x ", h->h[i]);
    }
    printf("\n");

    hipFree(cudaH);
    hipFree(cudaPwd);
    hipHostFree(h);
    hipHostFree(pwd);
    return 0;
}
