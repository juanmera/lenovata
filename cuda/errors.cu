#include <stdio.h>
#include "errors.cuh"

void handle_cuda_error(hipError_t err, const char *file, int line) {
    if (err != hipSuccess) {
        fprintf(stderr, "%s in %s at line %d\n",
            hipGetErrorString(err), file, line);
        if (err == hipErrorLaunchOutOfResources)
            fprintf(stderr, "Try decreasing THREADS in the corresponding hip/hip_runtime.h file. See doc/README-CUDA\n");
        exit(EXIT_FAILURE);
    }
}
